#include "hip/hip_runtime.h"
#define CUDA_KERNEL(func, ...) func<<<__VA_ARGS__>>>

#include <iostream>
#include "config.h"
#include "GPUExtension.h"

namespace GPUExtension
{
    template class GPUvector<aiVector3D>;
    template class GPUvector<Vertex>;

    // TODO : if in the future nvcc supports C++20, std::source_location, std::format and [[likely]] are better.
    static void CheckError(hipError_t error, const char* file, int line)
    {
        if (error == hipSuccess)
            return;
        std::cerr << "Cuda Error at file " << file << ", line : " << 
            line << " :" << hipGetErrorString(error);
        exit(EXIT_FAILURE);
    }

#define CHECK_ERROR(error) (CheckError((error), __FILE__, __LINE__))


    template<typename T>
    GPUvector<T>::GPUvector(T* init_CPUptr, size_t init_size): m_size(init_size), m_GPUptr(nullptr)
    {
        if (init_size == 0 || init_CPUptr == nullptr)
            return;
        size_t byteNum = m_size * sizeof(T);
        CHECK_ERROR(hipMalloc(&m_GPUptr, byteNum));
        CHECK_ERROR(hipMemcpy(m_GPUptr, init_CPUptr, byteNum, hipMemcpyHostToDevice));
        return;
    };
    
    template<typename T>
    GPUvector<T>::GPUvector(size_t init_size) : m_size(init_size), m_GPUptr(nullptr)
    {
        if (init_size == 0)
            return;
        size_t byteNum = m_size * sizeof(T);
        CHECK_ERROR(hipMalloc(&m_GPUptr, byteNum));
        return;
    };


    template<typename T>
    void GPUvector<T>::ToCPUVector(T* CPUptr)
    {
        CHECK_ERROR(hipMemcpy(CPUptr, m_GPUptr, m_size * sizeof(T), hipMemcpyDeviceToHost));
        return;
    }

    template<typename T>
    GPUvector<T>::~GPUvector()
    {
        CHECK_ERROR(hipFree(m_GPUptr));
        return;
    }

    void GPUSynchronize()
    {
        hipDeviceSynchronize();
        return;
    };

    namespace Mesh
    {
        __global__ void GPULoadVertices(size_t size, Vertex* dstVerts, aiVector3D* srcVerts, 
            aiVector3D* srcNorms, aiVector3D* srcTextures)
        {
            // size_t blockID = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z  * gridDim.y * gridDim.x;
            // size_t threadID = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
            // size_t id = blockID * blockDim.x * blockDim.y + threadID;
            size_t id = threadIdx.x + blockDim.x * blockIdx.x;
            if (id > size)
                return;
            Vertex& currVert = dstVerts[id];
            currVert.position.x = srcVerts[id].x;
            currVert.position.y = srcVerts[id].y;
            currVert.position.z = srcVerts[id].z;

            currVert.normalCoord.x = srcNorms[id].x;
            currVert.normalCoord.y = srcNorms[id].y;
            currVert.normalCoord.z = srcNorms[id].z;

            currVert.textureCoord.x = srcTextures[id].x;
            currVert.textureCoord.y = srcTextures[id].y;
            return;
        }

        __global__ void GPULoadVertices_OnlyVerts(size_t size, Vertex* dstVerts, aiVector3D* srcVerts)
        {
            // size_t blockID = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z  * gridDim.y * gridDim.x;
            // size_t threadID = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
            // size_t id = blockID * blockDim.x * blockDim.y + threadID;            
            size_t id = threadIdx.x + blockDim.x * blockIdx.x;
            if (id > size)
                return;
            Vertex& currVert = dstVerts[id];
            currVert.position.x = srcVerts[id].x;
            currVert.position.y = srcVerts[id].y;
            currVert.position.z = srcVerts[id].z;
            
            currVert.normalCoord.x = 0.0f;
            currVert.normalCoord.y = 0.0f;
            currVert.normalCoord.z = 0.0f;

            currVert.textureCoord.x = 0.0f;
            currVert.textureCoord.y = 0.0f;
            return;
        }

        __global__ void GPULoadVertices_NoTextures(size_t size, Vertex* dstVerts, aiVector3D* srcVerts, aiVector3D* srcNorms)
        {
            // size_t blockID = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z  * gridDim.y * gridDim.x;
            // size_t threadID = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
            // size_t id = blockID * blockDim.x * blockDim.y + threadID;      
            size_t id = threadIdx.x + blockDim.x * blockIdx.x;
            if (id > size)
                return;
            Vertex& currVert = dstVerts[id];
            currVert.position.x = srcVerts[id].x;
            currVert.position.y = srcVerts[id].y;
            currVert.position.z = srcVerts[id].z;

            currVert.normalCoord.x = srcNorms[id].x;
            currVert.normalCoord.y = srcNorms[id].y;
            currVert.normalCoord.z = srcNorms[id].z;

            currVert.textureCoord.x = 0.0f;
            currVert.textureCoord.y = 0.0f;
            return;
        }

        __global__ void GPULoadVertices_NoNorms(size_t size, Vertex* dstVerts, aiVector3D* srcVerts, aiVector3D* srcTextures)
        {
            // size_t blockID = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z  * gridDim.y * gridDim.x;
            // size_t threadID = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
            // size_t id = blockID * blockDim.x * blockDim.y + threadID;  
            size_t id = threadIdx.x + blockDim.x * blockIdx.x;
            if (id > size)
                return;
            Vertex& currVert = dstVerts[id];
            currVert.position.x = srcVerts[id].x;
            currVert.position.y = srcVerts[id].y;
            currVert.position.z = srcVerts[id].z;

            currVert.normalCoord.x = 0.0f;
            currVert.normalCoord.y = 0.0f;
            currVert.normalCoord.z = 0.0f;

            currVert.textureCoord.x = srcTextures[id].x;
            currVert.textureCoord.y = srcTextures[id].y;
            return;
        }

        void LoadVertices(GPUExecuteConfig& config, GPUvector<Vertex>& dstVerts,
            GPUvector<aiVector3D>& srcVerts, GPUvector<aiVector3D>& srcNorms, GPUvector<aiVector3D>& srcTextures)
        {
            dim3 threadsPerBlock{ config.m_threadsPerBlock[0], config.m_threadsPerBlock[1], config.m_threadsPerBlock[2] };
            dim3 blocksPerGrid{ config.m_blocksPerGrid[0], config.m_blocksPerGrid[1], config.m_blocksPerGrid[2] };

            bool hasNorms = srcNorms.size() != 0, hasTextures = srcTextures.size() != 0;
            if (hasNorms && hasTextures)
            {
                CUDA_KERNEL(GPULoadVertices, threadsPerBlock, blocksPerGrid)(dstVerts.size(), dstVerts.dataSymbol(),
                    srcVerts.dataSymbol(), srcNorms.dataSymbol(), srcTextures.dataSymbol());
            }
            else if (hasNorms && !hasTextures)
            {
                CUDA_KERNEL(GPULoadVertices_NoTextures, threadsPerBlock, blocksPerGrid)(dstVerts.size(), 
                    dstVerts.dataSymbol(), srcVerts.dataSymbol(), srcNorms.dataSymbol());
            }
            else if (!hasNorms && hasTextures)
            {
                CUDA_KERNEL(GPULoadVertices_NoNorms, threadsPerBlock, blocksPerGrid)(dstVerts.size(),
                    dstVerts.dataSymbol(), srcVerts.dataSymbol(), srcTextures.dataSymbol());
            }
            else
            {
                CUDA_KERNEL(GPULoadVertices_OnlyVerts, threadsPerBlock, blocksPerGrid)(dstVerts.size(),
                    dstVerts.dataSymbol(), srcVerts.dataSymbol());
            }

            if(config.m_needSynchronization)
            {
                GPUSynchronize();
            }
        }
    }
}